#include "spmm_cusparse.h"

void SpMMCuSparse::preprocess(float *vin, float *vout)
{
    hipsparseCreate(&handle);
    hipsparseCreateCsr(&matA, kNumV, kNumV, kNumE,
        d_ptr, d_idx, d_val,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    // Create dense matrix B
    hipsparseCreateDnMat(&matB, kNumV, kLen, kLen, vin,
            HIP_R_32F, HIPSPARSE_ORDER_ROW);
    // Create dense matrix C
    hipsparseCreateDnMat(&matC, kNumV, kLen, kLen, vout,
            HIP_R_32F, HIPSPARSE_ORDER_ROW);
    size_t bufferSize = 0;
// allocate an external buffer if needed
    hipsparseSpMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    checkCudaErrors(cudaMalloc2((void**)&buf, bufferSize));
}

void SpMMCuSparse::run(float *vin, float *vout)
{
    hipsparseSpMM(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, buf);
}