#include "gtest/gtest.h"
#include "util.h"
#include "valid.h"
#include "spmm_ref.h"
#include "spmm_opt.h"
#include "spmm_cusparse.h"

class SpMMTest:public testing::Test
{
protected:
    vector<void*> tensor_ptr;
    float *p_in_feat_vec, *p_out_feat_vec, *p_out_feat_vec_ref, *p_value;
    const int times = 10;
    CSR *g;
    virtual void SetUp()
    {
        p_in_feat_vec = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_out_feat_vec = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_out_feat_vec_ref = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_value = allocate<float>(kNumE, &tensor_ptr);
        g = new CSR(kNumV, kNumE, gptr, gidx, p_value);
    }
    virtual void TearDown()
    {
        for (auto item : tensor_ptr)
        {
            hipFree(item);
        }
    }
};

TEST_F(SpMMTest, validation)
{
    SpMMRef * spmmer_ref = new SpMMRef(g, kLen);
    // SpMMCuSparse * spmmer = new SpMMCuSparse(g, kLen);
    SpMMOpt * spmmer = new SpMMOpt(g, kLen);
    spmmer_ref->preprocess(p_in_feat_vec, p_out_feat_vec_ref);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    checkCudaErrors(hipMemset(p_out_feat_vec, 0, sizeof(float) * kNumV * kLen));
    checkCudaErrors(hipMemset(p_out_feat_vec_ref, 0, sizeof(float) * kNumV * kLen));
    spmmer_ref->run(p_in_feat_vec, p_out_feat_vec_ref);
    spmmer->run(p_in_feat_vec, p_out_feat_vec);
    checkCudaErrors(hipDeviceSynchronize());
    ASSERT_LT(valid(p_out_feat_vec, p_out_feat_vec_ref, kNumV * kLen), kNumV * kLen / 10000 + 1); 
}

TEST_F(SpMMTest, cusparse_performance)
{
    SpMMCuSparse * spmmer = new SpMMCuSparse(g, kLen);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    // warmup
    for (int i = 0; i < times; ++i)
    {
        spmmer->run(p_in_feat_vec, p_out_feat_vec);
    }
    double measured_time = 0;
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < times; ++i)
    {
        timestamp(t0);
        spmmer->run(p_in_feat_vec, p_out_feat_vec);
        checkCudaErrors(hipDeviceSynchronize());
        timestamp(t1);
        measured_time += getDuration(t0, t1);
    }
    dbg(measured_time / times);
}

TEST_F(SpMMTest, opt_performance)
{
    SpMMOpt * spmmer = new SpMMOpt(g, kLen);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    // warmup
    for (int i = 0; i < times; ++i)
    {
        spmmer->run(p_in_feat_vec, p_out_feat_vec);
    }
    double measured_time = 0;
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < times; ++i)
    {
        timestamp(t0);
        spmmer->run(p_in_feat_vec, p_out_feat_vec);
        checkCudaErrors(hipDeviceSynchronize());
        timestamp(t1);
        measured_time += getDuration(t0, t1);
    }
    dbg(measured_time / times);
}
